#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
	std::cout << "CUDA Vector Addition" << std::endl;
    int N = 10000;
    size_t size = N * sizeof(float);

	// Allocate host memory
	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);
	float* h_C = (float*)malloc(size);

	// Initialize input vectors
	for (int i = 0; i < N; i++) {
		h_A[i] = static_cast<float>(i);
		h_B[i] = static_cast<float>(i * 2);
	}

	// Allocate device memory
	float* d_A, * d_B, * d_C;
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	// Copy data from host to device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Launch kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);

	// Check for errors in kernel launch
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Synchronize device
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to synchronize after kernel launch (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy result from device to host
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Verify result
	for (int i = 0; i < N; i++) {
		if (h_C[i] != h_A[i] + h_B[i]) {
			fprintf(stderr, "Error: h_C[%d] = %f, expected %f\n", i, h_C[i], h_A[i] + h_B[i]);
			exit(EXIT_FAILURE);
		}
	}

	std::cout << "Vector addition completed successfully!" << std::endl;


	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}